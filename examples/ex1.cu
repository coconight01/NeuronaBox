#include "hip/hip_runtime.h"
#include "mpi.h"
#include "nccl.h"
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>

#define MPICHECK(cmd)                                                          \
  do {                                                                         \
    int e = cmd;                                                               \
    if (e != MPI_SUCCESS) {                                                    \
      printf("Failed: MPI error %s:%d '%d'\n", __FILE__, __LINE__, e);         \
      exit(EXIT_FAILURE);                                                      \
    }                                                                          \
  } while (0)

#define CUDACHECK(cmd)                                                         \
  do {                                                                         \
    hipError_t e = cmd;                                                       \
    if (e != hipSuccess) {                                                    \
      printf("Failed: Cuda error %s:%d '%s'\n", __FILE__, __LINE__,            \
             hipGetErrorString(e));                                           \
      exit(EXIT_FAILURE);                                                      \
    }                                                                          \
  } while (0)

#define NCCLCHECK(cmd)                                                         \
  do {                                                                         \
    ncclResult_t r = cmd;                                                      \
    if (r != ncclSuccess) {                                                    \
      printf("Failed, NCCL error %s:%d '%s'\n", __FILE__, __LINE__,            \
             ncclGetErrorString(r));                                           \
      exit(EXIT_FAILURE);                                                      \
    }                                                                          \
  } while (0)

static uint64_t getHostHash(const char *string) {
  // Based on DJB2a, result = result * 33 ^ char
  uint64_t result = 5381;
  for (int c = 0; string[c] != '\0'; c++) {
    result = ((result << 5) + result) ^ string[c];
  }
  return result;
}

static void getHostName(char *hostname, int maxlen) {
  gethostname(hostname, maxlen);
  for (int i = 0; i < maxlen; i++) {
    if (hostname[i] == '.') {
      hostname[i] = '\0';
      return;
    }
  }
}

extern char **environ;
int main(int argc, char *argv[]) {
  int i = 0;
  while (environ[i]) {
    if (environ[i][0] == 'N') {
      printf("%s\n", environ[i]);
    }
    i++;
  }
  int size = 32;

  int myRank, nRanks, localRank = 0;

  // initializing MPI
  MPICHECK(MPI_Init(&argc, &argv));
  MPICHECK(MPI_Comm_rank(MPI_COMM_WORLD, &myRank));
  MPICHECK(MPI_Comm_size(MPI_COMM_WORLD, &nRanks));

  // calculating localRank based on hostname which is used in selecting a GPU
  uint64_t hostHashs[nRanks];
  char hostname[1024];
  getHostName(hostname, 1024);
  hostHashs[myRank] = getHostHash(hostname);

  printf("myrank %d nranks %d hostname %s\n", myRank, nRanks, hostname);
  localRank = 0;

  ncclUniqueId id;
  ncclComm_t comm;
  float *sendbuff, *recvbuff;
  hipStream_t s;

  // get NCCL unique ID at rank 0 and broadcast it to all others
  if (myRank == 0)
    ncclGetUniqueId(&id);
  MPICHECK(MPI_Bcast((void *)&id, sizeof(id), MPI_BYTE, 0, MPI_COMM_WORLD));

  printf("get id at rank %d \n", myRank);
  // picking a GPU based on localRank, allocate device buffers
  CUDACHECK(hipSetDevice(localRank));
  CUDACHECK(hipMallocManaged(&sendbuff, size * sizeof(float)));
  CUDACHECK(hipMallocManaged(&recvbuff, size * sizeof(float)));
  CUDACHECK(hipStreamCreate(&s));

  // initializing NCCL
  NCCLCHECK(ncclCommInitRank(&comm, nRanks, id, myRank));
  sendbuff[0] = myRank + 1;
  // communicating using NCCL
  printf("before all reduce called\n");
  printf("send[0] %f at rank %d\n", sendbuff[0], myRank);
  NCCLCHECK(ncclAllReduce((const void *)sendbuff, (void *)recvbuff, size,
                          ncclFloat, ncclSum, comm, s));

  // completing NCCL operation by synchronizing on the CUDA stream
  CUDACHECK(hipStreamSynchronize(s));

  printf("after all reduce synced\n");
  printf("recv[0] %f\n", recvbuff[0]);
  // free device buffers
  CUDACHECK(hipFree(sendbuff));
  CUDACHECK(hipFree(recvbuff));

  // finalizing NCCL
  ncclCommDestroy(comm);

  // finalizing MPI
  MPICHECK(MPI_Finalize());

  printf("[MPI Rank %d] Success \n", myRank);
  return 0;
}